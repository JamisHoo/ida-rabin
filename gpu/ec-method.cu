#include "hip/hip_runtime.h"
/*
  Copyright (c) 2012-2014 DataLab, s.l. <http://www.datalab.es>
  This file is part of GlusterFS.

  This file is licensed to you under your choice of the GNU Lesser
  General Public License, version 3 or any later version (LGPLv3 or
  later), or the GNU General Public License, version 2 (GPLv2), in all
  cases as published by the Free Software Foundation.
*/
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <pthread.h>
#include "ec-method.h"

//There will be unknown bug if we put these two arrays in normal GPU memory.
__constant__ uint32_t GfPow_cuda[EC_GF_SIZE << 1]={1,2,4,8,16,32,64,128,29,58,116,232,205,135,19,38,76,152,45,90,180,117,234,201,143,3,6,12,24,48,96,192,157,39,78,156,37,74,148,53,106,212,181,119,238,193,159,35,70,140,5,10,20,40,80,160,93,186,105,210,185,111,222,161,95,190,97,194,153,47,94,188,101,202,137,15,30,60,120,240,253,231,211,187,107,214,177,127,254,225,223,163,91,182,113,226,217,175,67,134,17,34,68,136,13,26,52,104,208,189,103,206,129,31,62,124,248,237,199,147,59,118,236,197,151,51,102,204,133,23,46,92,184,109,218,169,79,158,33,66,132,21,42,84,168,77,154,41,82,164,85,170,73,146,57,114,228,213,183,115,230,209,191,99,198,145,63,126,252,229,215,179,123,246,241,255,227,219,171,75,150,49,98,196,149,55,110,220,165,87,174,65,130,25,50,100,200,141,7,14,28,56,112,224,221,167,83,166,81,162,89,178,121,242,249,239,195,155,43,86,172,69,138,9,18,36,72,144,61,122,244,245,247,243,251,235,203,139,11,22,44,88,176,125,250,233,207,131,27,54,108,216,173,71,142,1,2,4,8,16,32,64,128,29,58,116,232,205,135,19,38,76,152,45,90,180,117,234,201,143,3,6,12,24,48,96,192,157,39,78,156,37,74,148,53,106,212,181,119,238,193,159,35,70,140,5,10,20,40,80,160,93,186,105,210,185,111,222,161,95,190,97,194,153,47,94,188,101,202,137,15,30,60,120,240,253,231,211,187,107,214,177,127,254,225,223,163,91,182,113,226,217,175,67,134,17,34,68,136,13,26,52,104,208,189,103,206,129,31,62,124,248,237,199,147,59,118,236,197,151,51,102,204,133,23,46,92,184,109,218,169,79,158,33,66,132,21,42,84,168,77,154,41,82,164,85,170,73,146,57,114,228,213,183,115,230,209,191,99,198,145,63,126,252,229,215,179,123,246,241,255,227,219,171,75,150,49,98,196,149,55,110,220,165,87,174,65,130,25,50,100,200,141,7,14,28,56,112,224,221,167,83,166,81,162,89,178,121,242,249,239,195,155,43,86,172,69,138,9,18,36,72,144,61,122,244,245,247,243,251,235,203,139,11,22,44,88,176,125,250,233,207,131,27,54,108,216,173,71,142,1,0};
__constant__ uint32_t GfLog_cuda[EC_GF_SIZE << 1] = {256,255,1,25,2,50,26,198,3,223,51,238,27,104,199,75,4,100,224,14,52,141,239,129,28,193,105,248,200,8,76,113,5,138,101,47,225,36,15,33,53,147,142,218,240,18,130,69,29,181,194,125,106,39,249,185,201,154,9,120,77,228,114,166,6,191,139,98,102,221,48,253,226,152,37,179,16,145,34,136,54,208,148,206,143,150,219,189,241,210,19,92,131,56,70,64,30,66,182,163,195,72,126,110,107,58,40,84,250,133,186,61,202,94,155,159,10,21,121,43,78,212,229,172,115,243,167,87,7,112,192,247,140,128,99,13,103,74,222,237,49,197,254,24,227,165,153,119,38,184,180,124,17,68,146,217,35,32,137,46,55,63,209,91,149,188,207,205,144,135,151,178,220,252,190,97,242,86,211,171,20,42,93,158,132,60,57,83,71,109,65,162,31,45,67,216,183,123,164,118,196,23,73,236,127,12,111,246,108,161,59,82,41,157,85,170,251,96,134,177,187,204,62,90,203,89,95,176,156,169,160,81,11,245,22,235,122,117,44,215,79,174,213,233,230,231,173,232,116,214,244,234,168,80,88,175,255,1,25,2,50,26,198,3,223,51,238,27,104,199,75,4,100,224,14,52,141,239,129,28,193,105,248,200,8,76,113,5,138,101,47,225,36,15,33,53,147,142,218,240,18,130,69,29,181,194,125,106,39,249,185,201,154,9,120,77,228,114,166,6,191,139,98,102,221,48,253,226,152,37,179,16,145,34,136,54,208,148,206,143,150,219,189,241,210,19,92,131,56,70,64,30,66,182,163,195,72,126,110,107,58,40,84,250,133,186,61,202,94,155,159,10,21,121,43,78,212,229,172,115,243,167,87,7,112,192,247,140,128,99,13,103,74,222,237,49,197,254,24,227,165,153,119,38,184,180,124,17,68,146,217,35,32,137,46,55,63,209,91,149,188,207,205,144,135,151,178,220,252,190,97,242,86,211,171,20,42,93,158,132,60,57,83,71,109,65,162,31,45,67,216,183,123,164,118,196,23,73,236,127,12,111,246,108,161,59,82,41,157,85,170,251,96,134,177,187,204,62,90,203,89,95,176,156,169,160,81,11,245,22,235,122,117,44,215,79,174,213,233,230,231,173,232,116,214,244,234,168,80,88,175,0};

uint32_t GfPow[EC_GF_SIZE << 1]={1,2,4,8,16,32,64,128,29,58,116,232,205,135,19,38,76,152,45,90,180,117,234,201,143,3,6,12,24,48,96,192,157,39,78,156,37,74,148,53,106,212,181,119,238,193,159,35,70,140,5,10,20,40,80,160,93,186,105,210,185,111,222,161,95,190,97,194,153,47,94,188,101,202,137,15,30,60,120,240,253,231,211,187,107,214,177,127,254,225,223,163,91,182,113,226,217,175,67,134,17,34,68,136,13,26,52,104,208,189,103,206,129,31,62,124,248,237,199,147,59,118,236,197,151,51,102,204,133,23,46,92,184,109,218,169,79,158,33,66,132,21,42,84,168,77,154,41,82,164,85,170,73,146,57,114,228,213,183,115,230,209,191,99,198,145,63,126,252,229,215,179,123,246,241,255,227,219,171,75,150,49,98,196,149,55,110,220,165,87,174,65,130,25,50,100,200,141,7,14,28,56,112,224,221,167,83,166,81,162,89,178,121,242,249,239,195,155,43,86,172,69,138,9,18,36,72,144,61,122,244,245,247,243,251,235,203,139,11,22,44,88,176,125,250,233,207,131,27,54,108,216,173,71,142,1,2,4,8,16,32,64,128,29,58,116,232,205,135,19,38,76,152,45,90,180,117,234,201,143,3,6,12,24,48,96,192,157,39,78,156,37,74,148,53,106,212,181,119,238,193,159,35,70,140,5,10,20,40,80,160,93,186,105,210,185,111,222,161,95,190,97,194,153,47,94,188,101,202,137,15,30,60,120,240,253,231,211,187,107,214,177,127,254,225,223,163,91,182,113,226,217,175,67,134,17,34,68,136,13,26,52,104,208,189,103,206,129,31,62,124,248,237,199,147,59,118,236,197,151,51,102,204,133,23,46,92,184,109,218,169,79,158,33,66,132,21,42,84,168,77,154,41,82,164,85,170,73,146,57,114,228,213,183,115,230,209,191,99,198,145,63,126,252,229,215,179,123,246,241,255,227,219,171,75,150,49,98,196,149,55,110,220,165,87,174,65,130,25,50,100,200,141,7,14,28,56,112,224,221,167,83,166,81,162,89,178,121,242,249,239,195,155,43,86,172,69,138,9,18,36,72,144,61,122,244,245,247,243,251,235,203,139,11,22,44,88,176,125,250,233,207,131,27,54,108,216,173,71,142,1,0};
uint32_t GfLog[EC_GF_SIZE << 1] = {256,255,1,25,2,50,26,198,3,223,51,238,27,104,199,75,4,100,224,14,52,141,239,129,28,193,105,248,200,8,76,113,5,138,101,47,225,36,15,33,53,147,142,218,240,18,130,69,29,181,194,125,106,39,249,185,201,154,9,120,77,228,114,166,6,191,139,98,102,221,48,253,226,152,37,179,16,145,34,136,54,208,148,206,143,150,219,189,241,210,19,92,131,56,70,64,30,66,182,163,195,72,126,110,107,58,40,84,250,133,186,61,202,94,155,159,10,21,121,43,78,212,229,172,115,243,167,87,7,112,192,247,140,128,99,13,103,74,222,237,49,197,254,24,227,165,153,119,38,184,180,124,17,68,146,217,35,32,137,46,55,63,209,91,149,188,207,205,144,135,151,178,220,252,190,97,242,86,211,171,20,42,93,158,132,60,57,83,71,109,65,162,31,45,67,216,183,123,164,118,196,23,73,236,127,12,111,246,108,161,59,82,41,157,85,170,251,96,134,177,187,204,62,90,203,89,95,176,156,169,160,81,11,245,22,235,122,117,44,215,79,174,213,233,230,231,173,232,116,214,244,234,168,80,88,175,255,1,25,2,50,26,198,3,223,51,238,27,104,199,75,4,100,224,14,52,141,239,129,28,193,105,248,200,8,76,113,5,138,101,47,225,36,15,33,53,147,142,218,240,18,130,69,29,181,194,125,106,39,249,185,201,154,9,120,77,228,114,166,6,191,139,98,102,221,48,253,226,152,37,179,16,145,34,136,54,208,148,206,143,150,219,189,241,210,19,92,131,56,70,64,30,66,182,163,195,72,126,110,107,58,40,84,250,133,186,61,202,94,155,159,10,21,121,43,78,212,229,172,115,243,167,87,7,112,192,247,140,128,99,13,103,74,222,237,49,197,254,24,227,165,153,119,38,184,180,124,17,68,146,217,35,32,137,46,55,63,209,91,149,188,207,205,144,135,151,178,220,252,190,97,242,86,211,171,20,42,93,158,132,60,57,83,71,109,65,162,31,45,67,216,183,123,164,118,196,23,73,236,127,12,111,246,108,161,59,82,41,157,85,170,251,96,134,177,187,204,62,90,203,89,95,176,156,169,160,81,11,245,22,235,122,117,44,215,79,174,213,233,230,231,173,232,116,214,244,234,168,80,88,175,0};

static uint32_t ec_method_mul(uint32_t a, uint32_t b)
{
    if (a && b)
    {
        return GfPow[GfLog[a] + GfLog[b]];
    }

    return 0;
}

__device__ static uint32_t ec_method_div_cuda(uint32_t a, uint32_t b)
{
    if (b)
    {
        if (a)
        {
            return GfPow_cuda[EC_GF_SIZE - 1 + GfLog_cuda[a] - GfLog_cuda[b]];
        }
        return 0;
    }
    return EC_GF_SIZE;
}
static uint32_t ec_method_div(uint32_t a, uint32_t b)
{
    if (b)
    {
        if (a)
        {
            return GfPow[EC_GF_SIZE - 1 + GfLog[a] - GfLog[b]];
        }
        return 0;
    }
    return EC_GF_SIZE;
}

__global__ void ec_method_encode_kernel(uint32_t columns,uint32_t row,uint8_t* cuda_in,uint8_t *cuda_out,size_t total_trunk)
{
    uint32_t i;
    int trunk_id =(blockIdx.x * blockDim.x + threadIdx.x);
    if(trunk_id>=total_trunk)
        return;

    cuda_out += trunk_id*EC_METHOD_CHUNK_SIZE;
    cuda_in += trunk_id * columns * EC_METHOD_CHUNK_SIZE;

    ec_gf_muladd(0,cuda_out, cuda_in, EC_METHOD_WIDTH);
    cuda_in += EC_METHOD_CHUNK_SIZE;
    for (i = 1; i < columns; i++)
    {
        ec_gf_muladd(row,cuda_out,cuda_in, EC_METHOD_WIDTH);
        cuda_in += EC_METHOD_CHUNK_SIZE;
    }

}

size_t ec_method_encode(size_t size, uint32_t columns, uint32_t row,
                        uint8_t * in, uint8_t * out)
{
    
    uint32_t trunk = size /(EC_METHOD_CHUNK_SIZE * columns);
    uint8_t* cuda_in,*cuda_out;
    hipMalloc(&cuda_in,size);
    hipMalloc(&cuda_out,size/columns);

    hipMemcpy(cuda_in,in,size,hipMemcpyHostToDevice);

    int threadsPerBlock = 4;
    int blocksPerGrid = (trunk + threadsPerBlock - 1) / threadsPerBlock;

    ec_method_encode_kernel<<<blocksPerGrid,threadsPerBlock>>>(columns,row+1,cuda_in,cuda_out,trunk);
    //hipDeviceSynchronize();
    hipMemcpy(out,cuda_out,size/columns,hipMemcpyDeviceToHost);

    hipFree(cuda_in);
    hipFree(cuda_out);
    
    return size * EC_METHOD_CHUNK_SIZE;
}

void ec_method_test(){
    printf("Test\n");
}


__global__ void ec_method_decode_kernel(uint32_t columns,uint8_t * in,uint8_t *out,uint8_t *dummy,uint8_t *inv,size_t total_trunk)
{
    uint32_t i,j,last,value,tmp;
    int trunk_id = (blockIdx.x * blockDim.x + threadIdx.x);
    uint32_t inv_stride = EC_METHOD_MAX_FRAGMENTS+1;
    uint32_t in_stride = total_trunk*EC_METHOD_CHUNK_SIZE;
    if(trunk_id>=total_trunk)
        return;
    out+=trunk_id * EC_METHOD_CHUNK_SIZE *columns;

    for(i = 0;i<columns;i++)
    {
        last=0;
        j=0;
        do
        {
            while(j<columns && inv[i*inv_stride+j] == 0)
                j++;
            if(j<columns){
            	tmp = inv[i*inv_stride+j];
            	value = ec_method_div_cuda(last,tmp);
                last = tmp;
                ec_gf_muladd(value,out,in+j*in_stride+trunk_id*EC_METHOD_CHUNK_SIZE,EC_METHOD_WIDTH);
                j++;
            }
        }while(j<columns);
        ec_gf_muladd(last,out,dummy,EC_METHOD_WIDTH);
        out+=EC_METHOD_CHUNK_SIZE;
    }


}

size_t ec_method_decode(size_t size, uint32_t columns, uint32_t * rows,
                        uint8_t ** in, uint8_t * out)
{
    uint32_t i, j, k;
    uint32_t f;
    uint8_t **inv;
    uint8_t **mtx;
    uint8_t *dummy,*in_ptr;
    size /= EC_METHOD_CHUNK_SIZE;

    //Use some tricks to allocate 2-d array which is cache-friendly.
    inv = (uint8_t **)malloc(sizeof(uint8_t *) *EC_METHOD_MAX_FRAGMENTS);
    mtx = (uint8_t **)malloc(sizeof(uint8_t *) *EC_METHOD_MAX_FRAGMENTS);
    dummy =(uint8_t *)malloc(EC_METHOD_CHUNK_SIZE * sizeof(uint8_t));

    inv[0] = (uint8_t *)malloc((EC_METHOD_MAX_FRAGMENTS + 1)*EC_METHOD_MAX_FRAGMENTS * sizeof(uint8_t));
    mtx[0] = (uint8_t *)malloc(EC_METHOD_MAX_FRAGMENTS*EC_METHOD_MAX_FRAGMENTS * sizeof(uint8_t ));
    in_ptr = (uint8_t *)malloc(size* EC_METHOD_CHUNK_SIZE * columns);

    for(i=0;i<columns;i++)
    	memcpy(in_ptr+EC_METHOD_CHUNK_SIZE*size*i,in[i],EC_METHOD_CHUNK_SIZE*size);

    for(i=0;i<EC_METHOD_MAX_FRAGMENTS;i++)
        inv[i] = (*inv + (EC_METHOD_MAX_FRAGMENTS+1) * i),mtx[i]=(*mtx + EC_METHOD_MAX_FRAGMENTS * i);


    for(i=0;i<EC_METHOD_MAX_FRAGMENTS;i++){
        for(j=0;j<EC_METHOD_MAX_FRAGMENTS;j++)
            inv[i][j]=mtx[i][j]=0;
        inv[i][EC_METHOD_MAX_FRAGMENTS] = 0;
    }
    for(i=0;i<EC_METHOD_CHUNK_SIZE;i++)
        dummy[i]=0;

    for (i = 0; i < columns; i++)
    {
        inv[i][i] = 1;
        inv[i][columns] = 1;
    }
    for (i = 0; i < columns; i++)
    {
        mtx[i][columns - 1] = 1;
        for (j = columns - 1; j > 0; j--)
        {
            mtx[i][j - 1] = ec_method_mul(mtx[i][j], rows[i] + 1);
        }
    }

    for (i = 0; i < columns; i++)
    {
        f = mtx[i][i];
        for (j = 0; j < columns; j++)
        {
            mtx[i][j] = ec_method_div(mtx[i][j], f);
            inv[i][j] = ec_method_div(inv[i][j], f);
        }
        for (j = 0; j < columns; j++)
        {
            if (i != j)
            {
                f = mtx[j][i];
                for (k = 0; k < columns; k++)
                {
                    mtx[j][k] ^= ec_method_mul(mtx[i][k], f);
                    inv[j][k] ^= ec_method_mul(inv[i][k], f);
                }
            }
        }
    }
    
    uint8_t *cuda_in,*cuda_out,*cuda_dummy,*cuda_inv;


    hipMalloc(&cuda_in,sizeof(uint8_t )*columns*size*EC_METHOD_CHUNK_SIZE);
    hipMalloc(&cuda_out,size * EC_METHOD_CHUNK_SIZE * columns * sizeof(uint8_t));
    hipMalloc(&cuda_dummy,EC_METHOD_CHUNK_SIZE * sizeof(uint8_t));
    hipMalloc(&cuda_inv,sizeof(uint8_t) * EC_METHOD_MAX_FRAGMENTS *(EC_METHOD_MAX_FRAGMENTS+1));


    hipMemcpy(cuda_in,in_ptr,sizeof(uint8_t )*columns*size*EC_METHOD_CHUNK_SIZE,hipMemcpyHostToDevice);
    hipMemcpy(cuda_dummy,dummy,EC_METHOD_CHUNK_SIZE * sizeof(uint8_t),hipMemcpyHostToDevice);
    hipMemcpy(cuda_inv,*inv,sizeof(uint8_t) * EC_METHOD_MAX_FRAGMENTS *(EC_METHOD_MAX_FRAGMENTS+1),hipMemcpyHostToDevice);

    printf("Begin to decode\n");
	int threadsPerBlock = 4;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    ec_method_decode_kernel<<<blocksPerGrid,threadsPerBlock>>>(columns,cuda_in,cuda_out,cuda_dummy,cuda_inv,size);
    //ec_method_test_kernel<<<blocksPerGrid,threadsPerBlock>>>(cuda_out);
    //hipDeviceSynchronize();
    hipMemcpy(out,cuda_out,size * EC_METHOD_CHUNK_SIZE * columns * sizeof(uint8_t),hipMemcpyDeviceToHost);


    hipFree(cuda_in);
    hipFree(cuda_out);
    hipFree(cuda_dummy);
    hipFree(cuda_inv);

    free(dummy);
    free(inv[0]);
    free(mtx[0]);
    free(inv);
    free(mtx);

    return size * EC_METHOD_CHUNK_SIZE * columns;
}




